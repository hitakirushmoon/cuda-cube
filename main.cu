#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <algorithm>

#define stack_size_limit 9000000
using namespace std;

#include "consts.cuh"

template<typename T>
void read_file(const string &name, T *buffer) {
    std::ifstream is;
    is.open(name, ios::in | ios::binary | ios::ate);
    size_t filesize = is.tellg();
    cout << filesize / sizeof(T) << endl << name << endl;
    is.seekg(0, std::ios::beg);
    is.read((char *) buffer, filesize / 2);
    is.read((char *) buffer + filesize / 2, filesize - filesize / 2);
    is.close();
    is.clear();
    cout << "done" << endl;
}

class full_cube {
    constexpr static const uint8_t C[6][4] = {{0, 1, 2, 3},
                                              {4, 5, 6, 7},
                                              {0, 3, 4, 7},
                                              {6, 5, 2, 1},
                                              {4, 3, 2, 5},
                                              {6, 1, 0, 7}};;
    constexpr static const uint8_t CORNER_ROTATE[18][4] = {{3, 0, 1, 2},
                                                           {2, 3, 0, 1},
                                                           {1, 2, 3, 0},
                                                           {7, 4, 5, 6},
                                                           {6, 7, 4, 5},
                                                           {5, 6, 7, 4},
                                                           {7, 0, 3, 4},
                                                           {4, 7, 0, 3},
                                                           {3, 4, 7, 0},
                                                           {1, 6, 5, 2},
                                                           {2, 1, 6, 5},
                                                           {5, 2, 1, 6},
                                                           {5, 4, 3, 2},
                                                           {2, 5, 4, 3},
                                                           {3, 2, 5, 4},
                                                           {7, 6, 1, 0},
                                                           {0, 7, 6, 1},
                                                           {1, 0, 7, 6}};
    constexpr static const uint8_t CORNER_SYMS[48][8] = {{0, 1, 2, 3, 4, 5, 6, 7},
                                                         {3, 2, 1, 0, 7, 6, 5, 4},
                                                         {4, 5, 6, 7, 0, 1, 2, 3},
                                                         {7, 6, 5, 4, 3, 2, 1, 0},
                                                         {3, 0, 1, 2, 5, 6, 7, 4},
                                                         {0, 3, 2, 1, 6, 5, 4, 7},
                                                         {7, 4, 5, 6, 1, 2, 3, 0},
                                                         {4, 7, 6, 5, 2, 1, 0, 3},
                                                         {2, 3, 0, 1, 6, 7, 4, 5},
                                                         {1, 0, 3, 2, 5, 4, 7, 6},
                                                         {6, 7, 4, 5, 2, 3, 0, 1},
                                                         {5, 4, 7, 6, 1, 0, 3, 2},
                                                         {1, 2, 3, 0, 7, 4, 5, 6},
                                                         {2, 1, 0, 3, 4, 7, 6, 5},
                                                         {5, 6, 7, 4, 3, 0, 1, 2},
                                                         {6, 5, 4, 7, 0, 3, 2, 1},
                                                         {2, 5, 4, 3, 0, 7, 6, 1},
                                                         {1, 6, 7, 0, 3, 4, 5, 2},
                                                         {6, 1, 0, 7, 4, 3, 2, 5},
                                                         {5, 2, 3, 4, 7, 0, 1, 6},
                                                         {1, 6, 5, 2, 3, 4, 7, 0},
                                                         {2, 5, 6, 1, 0, 7, 4, 3},
                                                         {5, 2, 1, 6, 7, 0, 3, 4},
                                                         {6, 1, 2, 5, 4, 3, 0, 7},
                                                         {0, 7, 6, 1, 2, 5, 4, 3},
                                                         {3, 4, 5, 2, 1, 6, 7, 0},
                                                         {4, 3, 2, 5, 6, 1, 0, 7},
                                                         {7, 0, 1, 6, 5, 2, 3, 4},
                                                         {3, 4, 7, 0, 1, 6, 5, 2},
                                                         {0, 7, 4, 3, 2, 5, 6, 1},
                                                         {7, 0, 3, 4, 5, 2, 1, 6},
                                                         {4, 3, 0, 7, 6, 1, 2, 5},
                                                         {4, 7, 0, 3, 2, 1, 6, 5},
                                                         {7, 4, 3, 0, 1, 2, 5, 6},
                                                         {0, 3, 4, 7, 6, 5, 2, 1},
                                                         {3, 0, 7, 4, 5, 6, 1, 2},
                                                         {5, 4, 3, 2, 1, 0, 7, 6},
                                                         {6, 7, 0, 1, 2, 3, 4, 5},
                                                         {1, 0, 7, 6, 5, 4, 3, 2},
                                                         {2, 3, 4, 5, 6, 7, 0, 1},
                                                         {6, 5, 2, 1, 0, 3, 4, 7},
                                                         {5, 6, 1, 2, 3, 0, 7, 4},
                                                         {2, 1, 6, 5, 4, 7, 0, 3},
                                                         {1, 2, 5, 6, 7, 4, 3, 0},
                                                         {7, 6, 1, 0, 3, 2, 5, 4},
                                                         {4, 5, 2, 3, 0, 1, 6, 7},
                                                         {3, 2, 5, 4, 7, 6, 1, 0},
                                                         {0, 1, 6, 7, 4, 5, 2, 3}};
    constexpr static const uint8_t E[6][4] = {{0,  1,  2,  3},
                                              {11, 10, 9,  8},
                                              {0,  7,  8,  4},
                                              {2,  5,  10, 6},
                                              {3,  6,  11, 7},
                                              {1,  4,  9,  5}};
    constexpr static const uint8_t EDGE_ROTATE[18][4] = {{3,  0,  1,  2},
                                                         {2,  3,  0,  1},
                                                         {1,  2,  3,  0},
                                                         {8,  11, 10, 9},
                                                         {9,  8,  11, 10},
                                                         {10, 9,  8,  11},
                                                         {4,  0,  7,  8},
                                                         {8,  4,  0,  7},
                                                         {7,  8,  4,  0},
                                                         {6,  2,  5,  10},
                                                         {10, 6,  2,  5},
                                                         {5,  10, 6,  2},
                                                         {7,  3,  6,  11},
                                                         {11, 7,  3,  6},
                                                         {6,  11, 7,  3},
                                                         {5,  1,  4,  9},
                                                         {9,  5,  1,  4},
                                                         {4,  9,  5,  1}};
    constexpr static const uint8_t EDGE_SYMS[48][12] = {{0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11},
                                                        {0,  3,  2,  1,  7,  6,  5,  4,  8,  11, 10, 9},
                                                        {8,  11, 10, 9,  7,  6,  5,  4,  0,  3,  2,  1},
                                                        {8,  9,  10, 11, 4,  5,  6,  7,  0,  1,  2,  3},
                                                        {3,  0,  1,  2,  7,  4,  5,  6,  11, 8,  9,  10},
                                                        {1,  0,  3,  2,  4,  7,  6,  5,  9,  8,  11, 10},
                                                        {9,  8,  11, 10, 4,  7,  6,  5,  1,  0,  3,  2},
                                                        {11, 8,  9,  10, 7,  4,  5,  6,  3,  0,  1,  2},
                                                        {2,  3,  0,  1,  6,  7,  4,  5,  10, 11, 8,  9},
                                                        {2,  1,  0,  3,  5,  4,  7,  6,  10, 9,  8,  11},
                                                        {10, 9,  8,  11, 5,  4,  7,  6,  2,  1,  0,  3},
                                                        {10, 11, 8,  9,  6,  7,  4,  5,  2,  3,  0,  1},
                                                        {1,  2,  3,  0,  5,  6,  7,  4,  9,  10, 11, 8},
                                                        {3,  2,  1,  0,  6,  5,  4,  7,  11, 10, 9,  8},
                                                        {11, 10, 9,  8,  6,  5,  4,  7,  3,  2,  1,  0},
                                                        {9,  10, 11, 8,  5,  6,  7,  4,  1,  2,  3,  0},
                                                        {3,  6,  11, 7,  2,  10, 8,  0,  1,  5,  9,  4},
                                                        {1,  5,  9,  4,  2,  10, 8,  0,  3,  6,  11, 7},
                                                        {9,  5,  1,  4,  10, 2,  0,  8,  11, 6,  3,  7},
                                                        {11, 6,  3,  7,  10, 2,  0,  8,  9,  5,  1,  4},
                                                        {2,  5,  10, 6,  1,  9,  11, 3,  0,  4,  8,  7},
                                                        {2,  6,  10, 5,  3,  11, 9,  1,  0,  7,  8,  4},
                                                        {10, 6,  2,  5,  11, 3,  1,  9,  8,  7,  0,  4},
                                                        {10, 5,  2,  6,  9,  1,  3,  11, 8,  4,  0,  7},
                                                        {1,  4,  9,  5,  0,  8,  10, 2,  3,  7,  11, 6},
                                                        {3,  7,  11, 6,  0,  8,  10, 2,  1,  4,  9,  5},
                                                        {11, 7,  3,  6,  8,  0,  2,  10, 9,  4,  1,  5},
                                                        {9,  4,  1,  5,  8,  0,  2,  10, 11, 7,  3,  6},
                                                        {0,  7,  8,  4,  3,  11, 9,  1,  2,  6,  10, 5},
                                                        {0,  4,  8,  7,  1,  9,  11, 3,  2,  5,  10, 6},
                                                        {8,  4,  0,  7,  9,  1,  3,  11, 10, 5,  2,  6},
                                                        {8,  7,  0,  4,  11, 3,  1,  9,  10, 6,  2,  5},
                                                        {7,  8,  4,  0,  11, 9,  1,  3,  6,  10, 5,  2},
                                                        {4,  8,  7,  0,  9,  11, 3,  1,  5,  10, 6,  2},
                                                        {4,  0,  7,  8,  1,  3,  11, 9,  5,  2,  6,  10},
                                                        {7,  0,  4,  8,  3,  1,  9,  11, 6,  2,  5,  10},
                                                        {6,  11, 7,  3,  10, 8,  0,  2,  5,  9,  4,  1},
                                                        {5,  9,  4,  1,  10, 8,  0,  2,  6,  11, 7,  3},
                                                        {5,  1,  4,  9,  2,  0,  8,  10, 6,  3,  7,  11},
                                                        {6,  3,  7,  11, 2,  0,  8,  10, 5,  1,  4,  9},
                                                        {5,  10, 6,  2,  9,  11, 3,  1,  4,  8,  7,  0},
                                                        {6,  10, 5,  2,  11, 9,  1,  3,  7,  8,  4,  0},
                                                        {6,  2,  5,  10, 3,  1,  9,  11, 7,  0,  4,  8},
                                                        {5,  2,  6,  10, 1,  3,  11, 9,  4,  0,  7,  8},
                                                        {4,  9,  5,  1,  8,  10, 2,  0,  7,  11, 6,  3},
                                                        {7,  11, 6,  3,  8,  10, 2,  0,  4,  9,  5,  1},
                                                        {7,  3,  6,  11, 0,  2,  10, 8,  4,  1,  5,  9},
                                                        {4,  1,  5,  9,  0,  2,  10, 8,  7,  3,  6,  11}};

    uint8_t corner_perm[8] = {0, 1, 2, 3, 4, 5, 6, 7};
    uint8_t corner_or[8] = {};
    uint8_t edge_perm[12] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11};
    uint8_t edge_or[12] = {};
public:
    void print() {
        copy(begin(corner_or),
             end(corner_or),
             ostream_iterator<short>(cout, ", "));
        cout << endl;
        copy(begin(corner_perm),
             end(corner_perm),
             ostream_iterator<short>(cout, ", "));
        cout << endl;
        copy(begin(edge_or),
             end(edge_or),
             ostream_iterator<short>(cout, ", "));
        cout << endl;
        copy(begin(edge_perm),
             end(edge_perm),
             ostream_iterator<short>(cout, ", "));
        cout << endl;
        cout << endl;

    }

    full_cube &move(const uint32_t move) {
        if (move > 17) dual();
        uint32_t m = move / 3;
        uint8_t temp_perm[4];
        uint8_t temp_or[4];
        for (int i = 0; i < 4; i++) {
            temp_perm[i] = edge_perm[E[m][i]];
            temp_or[i] = edge_or[E[m][i]];
        }
        for (int i = 0; i < 4; i++) {
            edge_perm[EDGE_ROTATE[move][i]] = temp_perm[i];
            edge_or[EDGE_ROTATE[move][i]] = temp_or[i] ^ 1 ^ (move % 3 % 2);
        }

        for (int i = 0; i < 4; i++) {
            temp_perm[i] = corner_perm[C[m][i]];
            temp_or[i] = corner_or[C[m][i]];
        }
        for (int i = 0; i < 4; i++) {
            corner_perm[CORNER_ROTATE[move][i]] = temp_perm[i];
        }
        bool special_case = (move % 3 % 2 == 1 || m < 2);
        for (int i = 0; i < 4; i++) {
            corner_or[CORNER_ROTATE[move][i]] = special_case ? temp_or[i] : (temp_or[i] + (i % 2 + 1) * (3 - m / 2)) % 3;
        }
        return *this;
    }

    full_cube &move(const string &moves) {
        istringstream iss(moves);
        string item;
        while (getline(iss, item, ' ')) {
            uint8_t next_move = get_move_int(item);
            if (next_move == 18) {
                cout << "yo wtf is " << item << endl;
            }
            move(next_move);
        }
        return *this;
    }

    full_cube &dual() {
        uint8_t temp_corner_perm[8];
        uint8_t temp_corner_or[8];
        uint8_t temp_edge_perm[12];
        uint8_t temp_edge_or[12];

        for (uint8_t i = 0; i < 8; i++) {
            temp_corner_perm[corner_perm[i]] = i;
            temp_corner_or[corner_perm[i]] = ((corner_or[i] * 2) % 3);
        }
        for (uint8_t i = 0; i < 12; i++) {
            temp_edge_perm[edge_perm[i]] = i;
            temp_edge_or[edge_perm[i]] = edge_or[i];
        }
        copy(begin(temp_corner_perm), end(temp_corner_perm), begin(corner_perm));
        copy(begin(temp_corner_or), end(temp_corner_or), begin(corner_or));
        copy(begin(temp_edge_perm), end(temp_edge_perm), begin(edge_perm));
        copy(begin(temp_edge_or), end(temp_edge_or), begin(edge_or));

        return *this;
    }

    full_cube &sym(const uint8_t sym_dual) {
        int sym = sym_dual % 48;
        uint8_t temp_edge_perm[12];
        uint8_t temp_edge_or[12];
        for (uint8_t i = 0; i < 12; i++) {
            temp_edge_perm[edge_perm[EDGE_SYMS[sym][i]]] = i;
            temp_edge_or[i] = edge_or[EDGE_SYMS[sym][i]];
        }
        for (uint8_t i = 0; i < 12; i++) {
            edge_perm[temp_edge_perm[EDGE_SYMS[sym][i]]] = i;
        }
        copy(begin(temp_edge_or), end(temp_edge_or), begin(edge_or));

        uint8_t temp_corner_perm[8];
        uint8_t temp_corner_or[8];
        int c_index = sym >> 4;
        int lr_index = sym % 2;
        for (uint8_t i = 0; i < 8; i++) {
            temp_corner_perm[corner_perm[CORNER_SYMS[sym][i]]] = i;
            if (lr_index == 1) {
                corner_or[CORNER_SYMS[sym][i]] *= 2;
            }
            if (c_index != 0) {
                temp_corner_or[i] = ((2 - i % 2) * c_index + corner_or[CORNER_SYMS[sym][i]]) % 3;
            } else {
                temp_corner_or[i] = corner_or[CORNER_SYMS[sym][i]] % 3;
            }
        }
        for (uint8_t i = 0; i < 8; i++) {
            corner_perm[temp_corner_perm[CORNER_SYMS[sym][i]]] = i;
        }
        if (c_index == 0) {
            copy(begin(temp_corner_or), end(temp_corner_or), begin(corner_or));
        } else {
            for (uint8_t i = 0; i < 8; i++) {
                uint8_t new_i = temp_corner_perm[CORNER_SYMS[sym][i]];
                corner_or[new_i] = ((1 + i % 2) * c_index + temp_corner_or[new_i]) % 3;
            }
        }
        return sym_dual < 48 ? *this : dual();
    }

    uint32_t edge_perm_hash() {
        return factorial_hash(edge_perm, 12);
    }

    uint16_t edge_or_hash() {
        return normal_hash(edge_or, 2, 12);
    }

    uint32_t full_corner_hash() {
        return factorial_hash(corner_perm, 8) * 2187 + normal_hash(corner_or, 3, 8);
    }
};

__device__ uint64_t *device_edge_perm_pattern_database;
__device__ uint64_t *device_full_corner_pattern_database;

__device__ uint32_t device_edge_perm_neighbours[10044410][18];

__device__ uint16_t device_edge_or_neighbours[2048][18];
__device__ uint16_t device_edge_or_symmetries[2048][48];
__device__ uint16_t device_edge_or_dual[10044410][4];

__device__ uint32_t device_full_corner_neighbours[1841970][18];
__device__ uint32_t device_full_corner_dual[1841970];

__constant__ uint8_t device_rotated_moves[18][48];
__constant__ uint8_t device_rotated_syms[48][48];
__constant__ uint8_t device_dual_sym[48];

static auto host_edge_perm_pattern_database = new uint64_t[335544315];
static auto host_full_corner_pattern_database = new uint64_t[235772160];

static auto host_edge_perm_indexes = new uint32_t[5022205];
static auto host_edge_perm_neighbours = new uint32_t[10044410][18];

static auto host_edge_or_neighbours = new uint16_t[2048][18];
static auto host_edge_or_symmetries = new uint16_t[2048][48];
static auto host_edge_or_dual = new uint16_t[10044410][4];

static auto host_full_corner_indexes = new uint32_t[1841970];
static auto host_full_corner_neighbours = new uint32_t[1841970][18];
static auto host_full_corner_dual = new uint32_t[1841970];

class index_cube {
public:
    uint32_t edge_perm;
    uint32_t full_corner;
    uint16_t edge_or;
    uint8_t perm_sym, corner_sym;

    __host__ explicit index_cube(full_cube c) {
        edge_or = c.edge_or_hash();
        for (perm_sym = 0; perm_sym < 96; perm_sym++) {
            full_cube clone = c;
            uint32_t temp_hash = clone.sym(perm_sym).edge_perm_hash();
            edge_perm = lower_bound(host_edge_perm_indexes, host_edge_perm_indexes + 5022205, temp_hash) - host_edge_perm_indexes;
            if (host_edge_perm_indexes[edge_perm] == temp_hash) break;

        }
        edge_perm = edge_perm * 2 + perm_sym / 48;
        perm_sym %= 48;
        for (corner_sym = 0; corner_sym < 48; corner_sym++) {
            full_cube clone = c;
            uint32_t temp_hash = clone.sym(corner_sym).full_corner_hash();
            full_corner = lower_bound(host_full_corner_indexes, host_full_corner_indexes + 1841970, temp_hash) - host_full_corner_indexes;
            if (host_full_corner_indexes[full_corner] == temp_hash) break;
        }

    }

    index_cube() = default;

    ~index_cube() = default;

    friend ostream &operator<<(ostream &os, const index_cube &cube) {
        os << "edge_perm: " << cube.edge_perm << " edge_or: " << cube.edge_or << " full_corner: " << cube.full_corner << " perm_sym: "
           << (short) cube.perm_sym
           << " corner_sym: "
           << (short) cube.corner_sym;
        return os;
    }

    __device__ index_cube device_move(uint8_t i, index_cube &next) const {
        next.edge_or = device_edge_or_neighbours[edge_or][i];
        int perm_move = device_rotated_moves[i][perm_sym];
        next.edge_perm = device_edge_perm_neighbours[edge_perm][perm_move] / 48;
        next.perm_sym = device_rotated_syms[perm_sym][device_edge_perm_neighbours[edge_perm][perm_move] % 48];
        int corner_move = device_rotated_moves[i][corner_sym];
        next.full_corner = device_full_corner_neighbours[full_corner][corner_move] / 48;
        next.corner_sym = device_rotated_syms[corner_sym][device_full_corner_neighbours[full_corner][corner_move] % 48];
        return next;
    }

    __host__ index_cube host_move(uint8_t i, index_cube &next) const {
        next.edge_or = host_edge_or_neighbours[edge_or][i];
        int perm_move = host_rotated_moves[i][perm_sym];
        next.edge_perm = host_edge_perm_neighbours[edge_perm][perm_move] / 48;
        next.perm_sym = host_rotated_syms[perm_sym][host_edge_perm_neighbours[edge_perm][perm_move] % 48];
        int corner_move = host_rotated_moves[i][corner_sym];
        next.full_corner = host_full_corner_neighbours[full_corner][corner_move] / 48;
        next.corner_sym = host_rotated_syms[corner_sym][host_full_corner_neighbours[full_corner][corner_move] % 48];
        return next;
    }

    static index_cube get_cube_from_moves(const string &moves) {
        index_cube i{};
        return i.move(moves);
    }

    __host__ index_cube move(const string &moves) const {
        istringstream iss(moves);
        string item;
        index_cube current = *this;
        while (getline(iss, item, ' ')) {
            index_cube temp{};
            current.host_move(get_move_int(item), temp);
            current = temp;
        }
        return current;
    }
};

class search_state {
public:

    friend ostream &operator<<(ostream &os, const search_state &state) {
        os << "i: " << state.i << " h_corner: " << (short) state.h_corner << " h_edge: " << (short) state.h_edge;
        return os;
    }

    index_cube i;
    uint8_t h_corner, h_edge;

    search_state() = default;

    __host__ explicit search_state(const full_cube cube) : i(cube) {
        h_edge = 0;
        index_cube current = i;
        int current_mod = read_from_edge_array(current);
        while (current.edge_perm + current.edge_or != 0) {
            for (int move = 0; move < 18; move++) {
                index_cube next{};
                current.host_move(move, next);
                if ((read_from_edge_array(next) + 1) % 3 == current_mod) {
                    current_mod = (current_mod + 2) % 3;
                    current = next;
                    h_edge++;
                    break;
                }
            }
        }
        h_corner = read_from_corner_array(i);
    }

    __device__ __host__ static uint8_t max(uint8_t a, uint8_t b) {
        return a > b ? a : b;
    }

    __host__ bool host_move(int move, search_state &next) const {
        i.host_move(move, next.i);
        if (next.i.edge_perm / 2 + next.i.full_corner + next.i.edge_or == 0)
            return true;
        uint16_t corner_corrected_edge_or = host_edge_or_symmetries[next.i.edge_or][next.i.corner_sym];
        uint16_t edge_perm_corrected_edge_or = host_edge_or_symmetries[next.i.edge_or][next.i.perm_sym];
        uint16_t *dual = host_edge_or_dual[next.i.edge_perm];
        uint16_t edge_perm_corrected_dual_edge_or = get_inverse_bits_from_lehmer_code(edge_perm_corrected_edge_or, dual[0], dual[1], dual[2],
                                                                                      dual[3]);
        uint16_t dual_edge_or = host_edge_or_symmetries[edge_perm_corrected_dual_edge_or][host_dual_sym[next.i.perm_sym]];
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][next.i.corner_sym];
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][host_full_corner_dual[next.i.full_corner] % 48];
        next.h_corner = max(host_read_from_corner_array(next.i.full_corner, corner_corrected_edge_or),
                            host_read_from_corner_array(host_full_corner_dual[next.i.full_corner] / 48, dual_edge_or));
        if (h_corner != 0 && h_corner - 1 > next.h_corner) next.h_corner = h_corner - 1;
        uint16_t next_h_edge = host_read_from_edge_array(next.i.edge_perm,
                                                         next.i.edge_perm % 2 == 0 ? edge_perm_corrected_edge_or : edge_perm_corrected_dual_edge_or);
        next.h_edge = h_edge + (next_h_edge + h_edge * 2 + 1) % 3 - 1;
        return false;
    }


    __device__ bool device_move(int move, search_state &next) const {
        i.device_move(move, next.i);
        if (next.i.edge_perm / 2 + next.i.full_corner + next.i.edge_or == 0)
            return true;
        uint16_t corner_corrected_edge_or = device_edge_or_symmetries[next.i.edge_or][next.i.corner_sym];
        uint16_t edge_perm_corrected_edge_or = device_edge_or_symmetries[next.i.edge_or][next.i.perm_sym];
        uint16_t *dual = device_edge_or_dual[next.i.edge_perm];
        uint16_t edge_perm_corrected_dual_edge_or = get_inverse_bits_from_lehmer_code(edge_perm_corrected_edge_or, dual[0], dual[1], dual[2],
                                                                                      dual[3]);
        uint16_t dual_edge_or = device_edge_or_symmetries[edge_perm_corrected_dual_edge_or][device_dual_sym[next.i.perm_sym]];
        dual_edge_or = device_edge_or_symmetries[dual_edge_or][next.i.corner_sym];
        dual_edge_or = device_edge_or_symmetries[dual_edge_or][device_full_corner_dual[next.i.full_corner] % 48];
        next.h_corner = max(device_read_from_corner_array(next.i.full_corner, corner_corrected_edge_or),
                            device_read_from_corner_array(device_full_corner_dual[next.i.full_corner] / 48, dual_edge_or));
        if (h_corner != 0 && h_corner - 1 > next.h_corner) next.h_corner = h_corner - 1;
        uint16_t next_h_edge = device_read_from_edge_array(next.i.edge_perm,
                                                           next.i.edge_perm % 2 == 0 ? edge_perm_corrected_edge_or
                                                                                     : edge_perm_corrected_dual_edge_or);
        next.h_edge = h_edge + (next_h_edge + h_edge * 2 + 1) % 3 - 1;
        return false;
    }


    __host__ __device__  uint8_t h() const {
        return max(h_edge, h_corner);
    }

    static uint8_t read_from_edge_array(const index_cube &c) {
        uint16_t edge_or = c.edge_or;
        edge_or = host_edge_or_symmetries[edge_or][c.perm_sym];
        if (c.edge_perm % 2 == 1) {
            uint16_t *dual = host_edge_or_dual[c.edge_perm];
            edge_or = get_inverse_bits_from_lehmer_code(edge_or, dual[0], dual[1], dual[2], dual[3]);
        }
        return host_read_from_edge_array(c.edge_perm, edge_or);
    }

    __host__ static uint8_t host_read_from_edge_array(const uint32_t edge_perm, const uint16_t edge_or) {
        return (host_edge_perm_pattern_database[edge_perm / 2 * 64 + edge_or / 32] >> (edge_or % 32 * 2)) & 0b11;
    }

    __device__ static uint8_t device_read_from_edge_array(const uint32_t edge_perm, const uint16_t edge_or) {
        return (device_edge_perm_pattern_database[edge_perm / 2 * 64 + edge_or / 32] >> (edge_or % 32 * 2)) & 0b11;
    }

    __host__ static uint8_t read_from_corner_array(const index_cube &c) {
        uint16_t temp_edge_or = host_edge_or_symmetries[c.edge_or][c.corner_sym];
        uint16_t *dual = host_edge_or_dual[c.edge_perm];
        uint16_t dual_edge_or = c.edge_or;
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][c.perm_sym];
        dual_edge_or = get_inverse_bits_from_lehmer_code(dual_edge_or, dual[0], dual[1], dual[2], dual[3]);
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][host_dual_sym[c.perm_sym]];
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][c.corner_sym];
        dual_edge_or = host_edge_or_symmetries[dual_edge_or][host_full_corner_dual[c.full_corner] % 48];
        return max(host_read_from_corner_array(c.full_corner, temp_edge_or),
                   host_read_from_corner_array(host_full_corner_dual[c.full_corner] / 48, dual_edge_or));
    }

    __host__ static uint8_t host_read_from_corner_array(const uint32_t full_corner, const uint16_t edge_or) {
        return (host_full_corner_pattern_database[full_corner * 128 + edge_or / 16] >> (edge_or % 16 * 4)) & 0b1111;
    }

    __device__ static uint8_t device_read_from_corner_array(const uint32_t full_corner, const uint16_t edge_or) {
        return (device_full_corner_pattern_database[full_corner * 128 + edge_or / 16] >> (edge_or % 16 * 4)) & 0b1111;
    }
};

class mem_transfer {
public:
    uint8_t head;
    uint8_t prev_move, prev_move_2;
    uint8_t h;
    index_cube i;
};

class ida_stack {
public:
    int8_t head;
    int8_t moves[20];
    search_state state_stack[20 + 1];

    __host__ __device__ ida_stack() : state_stack{}, head{0}, moves{} {
        for (int8_t &move: moves) {
            move = -1;
        }
    }

    __host__ __device__ search_state &current_state() {
        return state_stack[head];
    }

    __host__ __device__ search_state &next_state() {
        return state_stack[head + 1];
    };

    __host__ __device__ int8_t &current_move() {
        return moves[head];
    }

    __host__ __device__ bool check_unnecessary_move() {
        if (head > 0) {
            if (moves[head] / 6 == moves[head - 1] / 6) {
                return (moves[head] / 3 <= moves[head - 1] / 3) || (head > 1 && moves[head] / 6 == moves[head - 2] / 6);
            }
        }
        return false;
    }
};

__device__ mem_transfer device_stacks[stack_size_limit];
__device__ int8_t moves[20];
static auto host_stacks = vector<mem_transfer>();
__device__ volatile int32_t device_result;
static int32_t host_result;
static int8_t host_moves[20]{};

static ida_stack result;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void read_files() {


    hipMemcpyToSymbol(HIP_SYMBOL(device_rotated_moves), host_rotated_moves, 18 * 48 * sizeof(uint8_t));
    hipMemcpyToSymbol(HIP_SYMBOL(device_rotated_syms), host_rotated_syms, 48 * 48 * sizeof(uint8_t));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dual_sym), host_dual_sym, 48 * sizeof(uint8_t));

    uint64_t *device_edge_perm_pattern_database_address;
    read_file("../edge_perm_edge_or_pattern_database", host_edge_perm_pattern_database);
    gpuErrchk(hipMalloc((void **) &device_edge_perm_pattern_database_address, 335544315 * sizeof(uint64_t)));
    hipMemcpy(device_edge_perm_pattern_database_address, host_edge_perm_pattern_database, 335544315 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_edge_perm_pattern_database), &device_edge_perm_pattern_database_address, sizeof(uint64_t *));

    uint64_t *device_full_corner_pattern_database_address;
    read_file("../corner_edge_or_pattern_database", host_full_corner_pattern_database);
    gpuErrchk(hipMalloc((void **) &device_full_corner_pattern_database_address, 235772160 * sizeof(uint64_t)));
    hipMemcpy(device_full_corner_pattern_database_address, host_full_corner_pattern_database, 235772160 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_full_corner_pattern_database), &device_full_corner_pattern_database_address, sizeof(uint64_t *));

    read_file("../edge_perm_indexes", host_edge_perm_indexes);

    read_file("../edge_perm_neighbours", (uint32_t *) host_edge_perm_neighbours);
    hipMemcpyToSymbol(HIP_SYMBOL(device_edge_perm_neighbours), (uint32_t *) host_edge_perm_neighbours, 10044410 * 18 * sizeof(uint32_t));

    read_file("../edge_or_neighbours", (uint16_t *) host_edge_or_neighbours);
    hipMemcpyToSymbol(HIP_SYMBOL(device_edge_or_neighbours), (uint16_t *) host_edge_or_neighbours, 2048 * 18 * sizeof(uint16_t));

    read_file("../edge_or_symmetries", (uint16_t *) host_edge_or_symmetries);
    hipMemcpyToSymbol(HIP_SYMBOL(device_edge_or_symmetries), (uint16_t *) host_edge_or_symmetries, 2048 * 48 * sizeof(uint16_t));

    read_file("../edge_or_dual", (uint16_t *) host_edge_or_dual);
    hipMemcpyToSymbol(HIP_SYMBOL(device_edge_or_dual), (uint16_t *) host_edge_or_dual, 10044410 * 4 * sizeof(uint16_t));

    read_file("../corner_indexes", host_full_corner_indexes);

    read_file("../corner_neighbours", (uint32_t *) host_full_corner_neighbours);
    hipMemcpyToSymbol(HIP_SYMBOL(device_full_corner_neighbours), (uint32_t *) host_full_corner_neighbours, 1841970 * 18 * sizeof(uint32_t));

    read_file("../corner_dual", host_full_corner_dual);
    hipMemcpyToSymbol(HIP_SYMBOL(device_full_corner_dual), host_full_corner_dual, 1841970 * sizeof(uint32_t));
    cout << "import finished" << endl;
    gpuErrchk(hipMalloc((void **) &device_result, sizeof(uint32_t)));

}

__host__ static int generate_stack(const search_state init, const uint8_t bound, bool add_to_vector, int target_count = -1) {
    ida_stack ss{};
    ss.current_state() = init;
//    host_stacks.clear();
    int count = 0;
    while (ss.head >= 0) {
        if (++ss.current_move() == 18) {
            ss.current_move() = -1;
            ss.head--;
            continue;
        }
        if (ss.check_unnecessary_move()) continue;
        search_state &current = ss.current_state();
        search_state &next_state = ss.next_state();
        if (current.host_move(ss.current_move(), next_state)) {
            cout << count << endl;
            for (const auto &item: ss.moves) {
                if (item == -1) break;
                cout << s[item] << ' ';
            }
            cout << endl;
            return -1;
        }
        if (next_state.h_corner - 1 > current.h_corner) {
            current.h_corner = next_state.h_corner - 1;
        }
        if (++ss.head + next_state.h() > bound) {
            if (count++ == target_count) {
                result = ss;
                return -1;
            }
            if (add_to_vector) {
                mem_transfer mem{};
                mem.head = ss.head;
                mem.h = ss.current_state().h_corner * 16 + ss.current_state().h_edge;
                mem.prev_move_2 = ss.moves[ss.head - 2];
                mem.prev_move = ss.moves[ss.head - 1];
                mem.i = ss.current_state().i;
                host_stacks.push_back(mem);
            }
            ss.head--;
            continue;
        }
    }
    return count;
}

__global__ void iterative_solve(const int limit, const int bound) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ ida_stack shared_stacks[];
    ida_stack &ss = shared_stacks[threadIdx.x];
    if (tid < limit) {
        ss = ida_stack();
        ss.head = device_stacks[tid].head;
        ss.moves[ss.head - 1] = device_stacks[tid].prev_move;
        ss.moves[ss.head - 2] = device_stacks[tid].prev_move_2;
        ss.current_state().h_corner = device_stacks[tid].h / 16;
        ss.current_state().h_edge = device_stacks[tid].h % 16;
        ss.current_state().i = device_stacks[tid].i;
        const int original_layer = ss.head;
        while (ss.head >= original_layer) {
            if (device_result != -1) {
                return;
            }
            if (++ss.current_move() == 18) {
                ss.current_move() = -1;
                ss.head--;
                continue;
            }
            if (ss.check_unnecessary_move()) continue;
            search_state &current = ss.current_state();
            search_state &next_state = ss.next_state();
            if (current.device_move(ss.current_move(), next_state)) {
                if (device_result != -1) {
                    return;
                }
                device_result = tid;
                for (int i = original_layer; i < 20; i++) {
                    moves[i] = ss.moves[i];
                }
                return;
            }
            if (next_state.h_corner - 1 > current.h_corner) {
                current.h_corner = next_state.h_corner - 1;
            }
            if (++ss.head + next_state.h() > bound) {
                ss.head--;
                continue;
            }
        }
    }
}

struct unaryfn : std::unary_function<int, int> {
    __device__ __host__ int operator()(int i) const { return sizeof(ida_stack) * i; }

};

void solve_cube(full_cube &f) {
    search_state state(f);
    cout << state << endl;
    int bound = state.h();
    int size;
    do {
        cout << bound << endl;
        auto t1 = chrono::high_resolution_clock::now();
        size = generate_stack(state, bound, false);
        auto t2 = chrono::high_resolution_clock::now();
        chrono::duration<double, milli> ms_double = t2 - t1;
        cout << "time: " << ms_double.count() << " " << size << endl;
        if (size == -1) {
            return;
        }
        bound++;
    } while (size * 14 < stack_size_limit);
    cout << "cuda time! w/ ";
    int true_bound = bound - 1;
    size = generate_stack(state, true_bound, true);
    cout << size << " states" << endl;
    host_result = -1;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, iterative_solve, unaryfn());
    // Round up according to array size
    gridSize = (size + blockSize - 1) / blockSize;
    cout << gridSize << " " << blockSize << endl;
    cout << "starting nodes: " << size << endl;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_stacks), &host_stacks[0], size * sizeof(mem_transfer)))
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_result), &host_result, sizeof(int)))
    gpuErrchk(hipDeviceSynchronize());
    while (true) {
        std::cout << bound << std::endl;
        auto t1 = chrono::high_resolution_clock::now();
        iterative_solve<<<gridSize, blockSize, blockSize * sizeof(ida_stack)>>>(size, bound);
        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipMemcpyFromSymbol(&host_result, HIP_SYMBOL(device_result), sizeof(int));
        hipMemcpyFromSymbol(&host_moves, HIP_SYMBOL(moves), 20 * sizeof(int8_t));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        auto t2 = chrono::high_resolution_clock::now();
        if (host_result != -1) {
            generate_stack(state, true_bound, false, host_result);
            for (int i = 0; i < 20 && result.moves[i] != -1; i++) {
                cout << s[result.moves[i]] << " ";
            }
            for (int i = result.head; i < 20 && host_moves[i] != -1; i++) {
                cout << s[host_moves[i]] << " ";
            }
            cout << endl;
            chrono::duration<double, milli> ms_double = t2 - t1;
            cout << "time: " << ms_double.count() << endl;
            break;
        }
        chrono::duration<double, milli> ms_double = t2 - t1;
        cout << "time: " << ms_double.count() << endl;
        bound++;
    }

}


int main() {
//    int nDevices;
//
//    hipGetDeviceCount(&nDevices);
//    for (int i = 0; i < nDevices; i++) {
//        hipDeviceProp_t prop;
//        hipGetDeviceProperties(&prop, i);
//        printf("%d.%d\n", prop.major, prop.minor);
//        printf("%d %d %d \n", prop.maxThreadsPerMultiProcessor, prop.maxThreadsPerBlock, prop.maxBlocksPerMultiProcessor);
//        printf("Device Number: %d\n", i);
//        printf("  Device name: %s\n", prop.name);
//        printf("  Memory Clock Rate (KHz): %d\n",
//               prop.memoryClockRate);
//        printf("  Memory Bus Width (bits): %d\n",
//               prop.memoryBusWidth);
//        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
//               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
//    }

    read_files();

    int maxbytes = 98304; // 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(iterative_solve), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    string shuffle;
    while (true) {
        getline(cin, shuffle);
        if (shuffle == "done") break;
        full_cube c;
        c.move(shuffle).print();
        auto t1 = chrono::high_resolution_clock::now();
        solve_cube(c);
        auto t2 = chrono::high_resolution_clock::now();
        chrono::duration<double, milli> ms_double = t2 - t1;
        cout << "time: " << ms_double.count() << endl;
        host_stacks.clear();
    }
    return 0;
}